#include "hip/hip_runtime.h"
#include <math.h>
#include "main.h"
#include <iostream>

__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int software_design::run(int N){
  std::cout << "N is " << N << std::endl;
  if (N<=0) return -1;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize; 

  int device = -1;

  hipGetDevice(&device);

  hipMemPrefetchAsync(x, N*sizeof(float), device, NULL);

  hipMemPrefetchAsync(y, N*sizeof(float), device, NULL);

  add<<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));


  // Free memory
  hipFree(x);
  hipFree(y);

  return(maxError);
}
