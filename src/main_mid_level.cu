#include "hip/hip_runtime.h"
#include"main.h"
#include<iostream>
#include<cmath>
#include<unordered_map>
#include<vector>
#include<cstdio>
#include <stdio.h>
#include <string.h>
#include <ctime>
#include <hiprand/hiprand_kernel.h>
using std::unordered_map;
using std::vector;
using hhg::Tool;
using std::string;
using std::cout;
using std::endl;
using std::ends;
__global__
void cuda_calcS(float * col1, float * col2, int * results,int size,float originS){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	float *newOrder;
	newOrder = (float*) malloc (size*sizeof(float));
	float table[4] = {0,0,0,0};
	//random number generator initialize
	hiprandState state;
	unsigned int seed = index;
	hiprand_init(seed, 0, 0, &state);	
	for(int iterationTime = index; iterationTime <1000; iterationTime+=stride){
		for(int i = 0; i < size; i++){
			newOrder[i] = col2[i];
		}
		float tmp; 
                int randomIndex;
		for(int i = size-1; i > 0; i--){
			randomIndex = hiprand(&state)%(i+1);
			tmp = newOrder[randomIndex];
			newOrder[randomIndex] = newOrder[i];
			newOrder[i] = tmp;
		}
		float s = 0;
		for (int index1 = 0; index1 < size; index1 ++){
			for(int index2 = index1 + 1 ; index2 < size; index2++){
				float rx = abs(col1[index1] - col1[index2]);
				float ry = abs(newOrder[index1] - newOrder[index2]);
				table[0] = 0;table[1] = 0;table[2] = 0;table[3] = 0;
				for(int index3 = 0; index3 < size; index3++){
					if(index3 != index1 && index3 != index2){
						float dy = abs(newOrder[index1] - newOrder[index3]);
						float dx = abs(col1[index1] - col1[index3]);
						if(dx <= rx){
							if(dy <= ry) table[0]++;
							else table[2]++;
						}
						else{
							if(dy <= ry) table[1]++;
							else table[3]++;
						}
					}
				}
				float a12 = table[1], a21 = table[2], a11 = table[0], a22 = table[3];
				float a1_ = a11 + a12, a2_ = a21 + a22, a_1 = a11 + a21, a_2 = a22 + a12;
				if( a1_==0 || a2_==0 || a_1==0 || a_2==0 ){
					continue;			
				}
				s += (size-2.0)*pow(a12*a21 - a11*a22 , 2) / (a_1*a_2*a1_*a2_);
			}
		}
		if(s >= originS) results[index] ++;
	}
	delete[] table;
	delete[] newOrder;
}
int main(int argc,char *argv[]){
	Tool *t = t->getInstance();
	std::clock_t start;
	if(argc != 2){
		cout<<"Invalid number of Parameters"<<endl;
		return 0;
	}
	int numofRow = atoi(argv[1]);
	double duration;
	unordered_map<string, vector<float>> data = t->dataGenerate(numofRow, 5, 1.0);
	start = std::clock();
	unordered_map<string, vector<float>> cols = t->randomlyPickColumns(data, 2);
	int size = numofRow;
	int blockSize = 256;
	int numBlocks = (1000 + blockSize - 1) / blockSize;
	float *col1, *col2;
	int *results;
	hipMallocManaged(&col1, size*sizeof(float));
	hipMallocManaged(&col2, size*sizeof(float));
	hipMallocManaged(&results, numBlocks*blockSize*sizeof(int));
	hipMemset(results, 0, numBlocks*blockSize*sizeof(int));
	int counter = 0;
	float originS = 0;
	originS = t->calcS(cols);
	//cout<<"Original S: "<< originS << endl << "====================================="<<endl;
	for(int i = 0 ; i < size ; i++){
		auto iter = data.begin();
		col1[i] = iter->second[i];
		iter++;
		col2[i] = iter->second[i];
	}
	int device = -1;
	hipGetDevice(&device);
	hipMemPrefetchAsync(col1, size*sizeof(float), device, NULL);
	hipMemPrefetchAsync(col2, size*sizeof(float), device, NULL);
	hipMemPrefetchAsync(results, numBlocks*blockSize*sizeof(int), device, NULL);
	hipMemPrefetchAsync(&size, sizeof(int), device, NULL);
	cuda_calcS<<<numBlocks, blockSize>>>(col1, col2, results, size, originS);
	hipDeviceSynchronize();
	for(int i = 0 ; i < numBlocks*blockSize; i++){
		counter+= results[i];
	}

	float p = counter / t->iterTime_;
	cout<<"There are "<<counter<<" S greater than the original S."<<endl;
	cout<<"P = "<<p<<endl;
	duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
	std::cout<< "duration: "<<duration <<"seconds\n";
	hipFree(col1);
	hipFree(col2);
	hipFree(results);
}
