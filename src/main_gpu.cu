#include "hip/hip_runtime.h"
#include"main.h"
#include<iostream>
#include<cmath>
#include<unordered_map>
#include<vector>
#include<cstdio>
#include <stdio.h>
#include <string.h>
#include <ctime>
using std::unordered_map;
using std::vector;
using hhg::Tool;
using std::string;
using std::cout;
using std::endl;
using std::ends;
__global__
void cuda_calcS(float * col1, float * col2, float * results,int size){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int index1 = index; index1 < size; index1 += stride){
		for(int index2 = index1 + 1 ; index2 < size; index2++){
			float rx = abs(col1[index1] - col1[index2]);
			float ry = abs(col2[index1] - col2[index2]);
			float table[4] = {0,0,0,0};
			for(int index3 = 0; index3 < size; index3++){
				if(index3 != index1 && index3 != index2){
					float dy = abs(col2[index1] - col2[index3]);
					float dx = abs(col1[index1] - col1[index3]);
					if(dx <= rx){
						if(dy <= ry) table[0]++;
						else table[2]++;
					}
					else{
						if(dy <= ry) table[1]++;
						else table[3]++;
					}
				}
			}
			float a12 = table[1], a21 = table[2], a11 = table[0], a22 = table[3];
			float a1_ = a11 + a12, a2_ = a21 + a22, a_1 = a11 + a21, a_2 = a22 + a12;
			//cout<<a11<<ends<<a12<<endl<<a21<<ends<<a22<<endl;
			if( a1_==0 || a2_==0 || a_1==0 || a_2==0 ){
				//results[index]--;
				continue;			
			}
			//cout<<size<<endl;
			/*if(index == 1)
				printf("Hello from block %d, thread %d, The table is %lf %lf %lf %lf\n", blockIdx.x, threadIdx.x, a11,a12,a21,a22);*/
			results[index] += (size-2.0)*pow(a12*a21 - a11*a22 , 2) / (a_1*a_2*a1_*a2_);
			delete[] table;
		}
	}
}
int main(int argc,char *argv[]){
	Tool *t = t->getInstance();
	std::clock_t start;
	if(argc != 2){
		cout<<"Invalid number of Parameters"<<endl;
		return 0;
	}
	int numofRow = atoi(argv[1]);
	double duration;
	unordered_map<string, vector<float>> data = t->dataGenerate(numofRow, 5, 1.0);//{{"Australia",{0,1,2,2,3,6,7}},{"U.S.",{5,3,2,6,2,3,4}}};//,{"France",{99,1,2,6,9,2,4}}};
	start = std::clock();
	unordered_map<string, vector<float>> cols = t->randomlyPickColumns(data, 2);
	int size = numofRow;
	int blockSize = 256;
	int numBlocks = (size + blockSize - 1) / blockSize;
	float *col1, *col2, *results;
	hipMallocManaged(&col1, size*sizeof(float));
	hipMallocManaged(&col2, size*sizeof(float));
	hipMallocManaged(&results, numBlocks*blockSize*sizeof(float));
	int counter = 0;
	float originS = 0;
	originS = t->calcS(cols);
	//cout<<"Original S: "<< originS << endl << "====================================="<<endl;

	float s;
	for(int num = 0; num < t->iterTime_; num ++){
		s = 0;
		unordered_map<string, vector<float>> newOrder = t->reorderData(cols);
		for(int i = 0 ; i < size ; i++){
			auto iter = newOrder.begin();
			col1[i] = iter->second[i];
			iter++;
			col2[i] = iter->second[i];
		}
		hipMemset(results, 0, numBlocks*blockSize*sizeof(float));
		int device = -1;
		hipGetDevice(&device);
		hipMemPrefetchAsync(col1, size*sizeof(float), device, NULL);
		hipMemPrefetchAsync(col2, size*sizeof(float), device, NULL);
		hipMemPrefetchAsync(results, numBlocks*blockSize*sizeof(float), device, NULL);
		hipMemPrefetchAsync(&size, sizeof(int), device, NULL);
		cuda_calcS<<<numBlocks, blockSize>>>(col1, col2, results,size);
		hipDeviceSynchronize();
		for(int i = 0 ; i < numBlocks*blockSize; i++){
			s+= results[i];
		}
		if(s >= originS) counter++;
		//cout<<"S: "<<s<<endl<<"====================================="<<endl;
	}
	
	float p = counter / t->iterTime_;
	cout<<"There are "<<counter<<" S greater than the original S."<<endl;
	cout<<"P = "<<p<<endl;
	duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
	std::cout<< duration <<'\n';
	hipFree(col1);
	hipFree(col2);
	hipFree(results);
}
